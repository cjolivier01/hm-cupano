#include "cudaMakeFull.h"
#include "cudaTypes.h"

#include "cudaUtils.cuh"

#include <hip/hip_runtime.h>
#include <cassert>

#if (CUDART_VERSION >= 11000)
#include <hip/hip_bf16.h>
#endif
#include <hip/hip_fp16.h>

namespace hm {
namespace cupano {
namespace cuda {

} // namespace cuda
} // namespace cupano
} // namespace hm
