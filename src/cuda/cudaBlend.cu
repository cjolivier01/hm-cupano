#include "hip/hip_runtime.h"
#include "cudaBlend.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <vector>

// =============================================================================
// Macro to check CUDA calls and return on error.
#define CUDA_CHECK(call)                                                                          \
  do {                                                                                            \
    hipError_t _err = (call);                                                                    \
    if (_err != hipSuccess) {                                                                    \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(_err)); \
      return _err;                                                                                \
    }                                                                                             \
  } while (0)

// =============================================================================
// Templated CUDA Kernels for Batched Laplacian Blending
// =============================================================================

// -----------------------------------------------------------------------------
// Fused downsample kernel for two RGB images.
// Instead of launching separate kernels for image1 and image2, we fuse them
// into one kernel call. Each output pixel is computed by averaging a 2x2 block.
// -----------------------------------------------------------------------------
template <typename T>
__global__ void FusedBatchedDownsampleKernelRGB(
    const T* __restrict__ input1,
    const T* __restrict__ input2,
    int inWidth,
    int inHeight,
    T* __restrict__ output1,
    T* __restrict__ output2,
    int outWidth,
    int outHeight,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= outWidth || y >= outHeight)
    return;

  int inImageSize = inWidth * inHeight * 3;
  int outImageSize = outWidth * outHeight * 3;
  const T* inImage1 = input1 + b * inImageSize;
  const T* inImage2 = input2 + b * inImageSize;
  T* outImage1 = output1 + b * outImageSize;
  T* outImage2 = output2 + b * outImageSize;

  int inX = x * 2;
  int inY = y * 2;
  float sumR1 = 0.0f, sumG1 = 0.0f, sumB1 = 0.0f;
  float sumR2 = 0.0f, sumG2 = 0.0f, sumB2 = 0.0f;
  int count = 0;
#pragma unroll
  for (int dy = 0; dy < 2; dy++) {
#pragma unroll
    for (int dx = 0; dx < 2; dx++) {
      int ix = inX + dx;
      int iy = inY + dy;
      if (ix < inWidth && iy < inHeight) {
        int idx = (iy * inWidth + ix) * 3;
        sumR1 += static_cast<float>(inImage1[idx + 0]);
        sumG1 += static_cast<float>(inImage1[idx + 1]);
        sumB1 += static_cast<float>(inImage1[idx + 2]);
        sumR2 += static_cast<float>(inImage2[idx + 0]);
        sumG2 += static_cast<float>(inImage2[idx + 1]);
        sumB2 += static_cast<float>(inImage2[idx + 2]);
        count++;
      }
    }
  }
  int outIdx = (y * outWidth + x) * 3;
  outImage1[outIdx + 0] = static_cast<T>(sumR1 / count);
  outImage1[outIdx + 1] = static_cast<T>(sumG1 / count);
  outImage1[outIdx + 2] = static_cast<T>(sumB1 / count);
  outImage2[outIdx + 0] = static_cast<T>(sumR2 / count);
  outImage2[outIdx + 1] = static_cast<T>(sumG2 / count);
  outImage2[outIdx + 2] = static_cast<T>(sumB2 / count);
}

// -----------------------------------------------------------------------------
// Batched downsample kernel for a single image with __restrict__ and unrolling.
// This kernel is kept in case separate launches are desired.
// -----------------------------------------------------------------------------
template <typename T>
__global__ void BatchedDownsampleKernelRGB(
    const T* __restrict__ input,
    int inWidth,
    int inHeight,
    T* __restrict__ output,
    int outWidth,
    int outHeight,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= outWidth || y >= outHeight)
    return;

  int inImageSize = inWidth * inHeight * 3;
  int outImageSize = outWidth * outHeight * 3;
  const T* inImage = input + b * inImageSize;
  T* outImage = output + b * outImageSize;

  int inX = x * 2;
  int inY = y * 2;
  float sumR = 0.0f, sumG = 0.0f, sumB = 0.0f;
  int count = 0;
#pragma unroll
  for (int dy = 0; dy < 2; dy++) {
#pragma unroll
    for (int dx = 0; dx < 2; dx++) {
      int ix = inX + dx;
      int iy = inY + dy;
      if (ix < inWidth && iy < inHeight) {
        int idx = (iy * inWidth + ix) * 3;
        sumR += static_cast<float>(inImage[idx + 0]);
        sumG += static_cast<float>(inImage[idx + 1]);
        sumB += static_cast<float>(inImage[idx + 2]);
        count++;
      }
    }
  }
  int outIdx = (y * outWidth + x) * 3;
  outImage[outIdx + 0] = static_cast<T>(sumR / count);
  outImage[outIdx + 1] = static_cast<T>(sumG / count);
  outImage[outIdx + 2] = static_cast<T>(sumB / count);
}

// -----------------------------------------------------------------------------
// Batched downsample kernel for a single-channel mask with __restrict__ and unrolling.
// -----------------------------------------------------------------------------
template <typename T>
__global__ void BatchedDownsampleKernelMask(
    const T* __restrict__ input,
    int inWidth,
    int inHeight,
    T* __restrict__ output,
    int outWidth,
    int outHeight) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= outWidth || y >= outHeight)
    return;

  int inX = x * 2;
  int inY = y * 2;
  float sum = 0.0f;
  int count = 0;
#pragma unroll
  for (int dy = 0; dy < 2; dy++) {
#pragma unroll
    for (int dx = 0; dx < 2; dx++) {
      int ix = inX + dx;
      int iy = inY + dy;
      if (ix < inWidth && iy < inHeight) {
        sum += static_cast<float>(input[iy * inWidth + ix]);
        count++;
      }
    }
  }
  output[y * outWidth + x] = static_cast<T>(sum / count);
}

// -----------------------------------------------------------------------------
// Batched upsample kernel for RGB images using bilinear interpolation.
// -----------------------------------------------------------------------------
template <typename T, typename F_T>
__global__ void BatchedUpsampleKernelRGB(
    const T* __restrict__ input,
    int inWidth,
    int inHeight,
    T* __restrict__ output,
    int outWidth,
    int outHeight,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= outWidth || y >= outHeight)
    return;

  int inImageSize = inWidth * inHeight * 3;
  int outImageSize = outWidth * outHeight * 3;
  const T* inImage = input + b * inImageSize;
  T* outImage = output + b * outImageSize;

  F_T gx = static_cast<F_T>(x) / 2.0f;
  F_T gy = static_cast<F_T>(y) / 2.0f;
  int gxi = floorf(gx);
  int gyi = floorf(gy);
  F_T dx = gx - gxi;
  F_T dy = gy - gyi;
  int gxi1 = min(gxi + 1, inWidth - 1);
  int gyi1 = min(gyi + 1, inHeight - 1);

  int idx00 = (gyi * inWidth + gxi) * 3;
  int idx10 = (gyi * inWidth + gxi1) * 3;
  int idx01 = (gyi1 * inWidth + gxi) * 3;
  int idx11 = (gyi1 * inWidth + gxi1) * 3;

  F_T outR, outG, outB;
  { // R channel interpolation.
    F_T val00 = static_cast<F_T>(inImage[idx00 + 0]);
    F_T val10 = static_cast<F_T>(inImage[idx10 + 0]);
    F_T val01 = static_cast<F_T>(inImage[idx01 + 0]);
    F_T val11 = static_cast<F_T>(inImage[idx11 + 0]);
    F_T val0 = val00 * (1.0f - dx) + val10 * dx;
    F_T val1 = val01 * (1.0f - dx) + val11 * dx;
    outR = val0 * (1.0f - dy) + val1 * dy;
  }
  { // G channel interpolation.
    F_T val00 = static_cast<F_T>(inImage[idx00 + 1]);
    F_T val10 = static_cast<F_T>(inImage[idx10 + 1]);
    F_T val01 = static_cast<F_T>(inImage[idx01 + 1]);
    F_T val11 = static_cast<F_T>(inImage[idx11 + 1]);
    F_T val0 = val00 * (1.0f - dx) + val10 * dx;
    F_T val1 = val01 * (1.0f - dx) + val11 * dx;
    outG = val0 * (1.0f - dy) + val1 * dy;
  }
  { // B channel interpolation.
    F_T val00 = static_cast<F_T>(inImage[idx00 + 2]);
    F_T val10 = static_cast<F_T>(inImage[idx10 + 2]);
    F_T val01 = static_cast<F_T>(inImage[idx01 + 2]);
    F_T val11 = static_cast<F_T>(inImage[idx11 + 2]);
    F_T val0 = val00 * (1.0f - dx) + val10 * dx;
    F_T val1 = val01 * (1.0f - dx) + val11 * dx;
    outB = val0 * (1.0f - dy) + val1 * dy;
  }
  int idxOut = (y * outWidth + x) * 3;
  outImage[idxOut + 0] = static_cast<T>(outR);
  outImage[idxOut + 1] = static_cast<T>(outG);
  outImage[idxOut + 2] = static_cast<T>(outB);
}

// -----------------------------------------------------------------------------
// Batched computation of the Laplacian for RGB images.
// For each pixel, compute Laplacian = Gaussian_high - upsample(Gaussian_low).
// -----------------------------------------------------------------------------
template <typename T, typename F_T>
__global__ void BatchedComputeLaplacianKernelRGB(
    const T* __restrict__ gaussHigh,
    int highWidth,
    int highHeight,
    const T* __restrict__ gaussLow,
    int lowWidth,
    int lowHeight,
    T* __restrict__ laplacian,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= highWidth || y >= highHeight)
    return;

  int highImageSize = highWidth * highHeight * 3;
  int lowImageSize = lowWidth * lowHeight * 3;
  const T* highImage = gaussHigh + b * highImageSize;
  const T* lowImage = gaussLow + b * lowImageSize;
  T* lapImage = laplacian + b * highImageSize;

  F_T gx = static_cast<F_T>(x) / 2.0f;
  F_T gy = static_cast<F_T>(y) / 2.0f;
  int gxi = floorf(gx);
  int gyi = floorf(gy);
  F_T dx = gx - static_cast<F_T>(gxi);
  F_T dy = gy - static_cast<F_T>(gyi);
  int gxi1 = min(gxi + 1, lowWidth - 1);
  int gyi1 = min(gyi + 1, lowHeight - 1);

  int idx00 = (gyi * lowWidth + gxi) * 3;
  int idx10 = (gyi * lowWidth + gxi1) * 3;
  int idx01 = (gyi1 * lowWidth + gxi) * 3;
  int idx11 = (gyi1 * lowWidth + gxi1) * 3;

  const F_T F_ONE = static_cast<F_T>(1.0);
  F_T upR, upG, upB;
  { // R channel.
    F_T val00 = static_cast<F_T>(lowImage[idx00 + 0]);
    F_T val10 = static_cast<F_T>(lowImage[idx10 + 0]);
    F_T val01 = static_cast<F_T>(lowImage[idx01 + 0]);
    F_T val11 = static_cast<F_T>(lowImage[idx11 + 0]);
    F_T val0 = val00 * (F_ONE - dx) + val10 * dx;
    F_T val1 = val01 * (F_ONE - dx) + val11 * dx;
    upR = val0 * (F_ONE - dy) + val1 * dy;
  }
  { // G channel.
    F_T val00 = static_cast<F_T>(lowImage[idx00 + 1]);
    F_T val10 = static_cast<F_T>(lowImage[idx10 + 1]);
    F_T val01 = static_cast<F_T>(lowImage[idx01 + 1]);
    F_T val11 = static_cast<F_T>(lowImage[idx11 + 1]);
    F_T val0 = val00 * (F_ONE - dx) + val10 * dx;
    F_T val1 = val01 * (F_ONE - dx) + val11 * dx;
    upG = val0 * (F_ONE - dy) + val1 * dy;
  }
  { // B channel.
    F_T val00 = static_cast<F_T>(lowImage[idx00 + 2]);
    F_T val10 = static_cast<F_T>(lowImage[idx10 + 2]);
    F_T val01 = static_cast<F_T>(lowImage[idx01 + 2]);
    F_T val11 = static_cast<F_T>(lowImage[idx11 + 2]);
    F_T val0 = val00 * (F_ONE - dx) + val10 * dx;
    F_T val1 = val01 * (F_ONE - dx) + val11 * dx;
    upB = val0 * (F_ONE - dy) + val1 * dy;
  }
  int idxHigh = (y * highWidth + x) * 3;
  lapImage[idxHigh + 0] = static_cast<T>(static_cast<F_T>(highImage[idxHigh + 0]) - upR);
  lapImage[idxHigh + 1] = static_cast<T>(static_cast<F_T>(highImage[idxHigh + 1]) - upG);
  lapImage[idxHigh + 2] = static_cast<T>(static_cast<F_T>(highImage[idxHigh + 2]) - upB);
}

// -----------------------------------------------------------------------------
// Batched blend kernel for RGB images.
// Blends two Laplacian pyramid levels using a shared single-channel mask.
// -----------------------------------------------------------------------------
template <typename T, typename F_T>
__global__ void BatchedBlendKernelRGB(
    const T* __restrict__ lap1,
    const T* __restrict__ lap2,
    const T* __restrict__ mask,
    T* __restrict__ blended,
    int width,
    int height,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int imageSizeRGB = width * height * 3;
  const T* lap1Image = lap1 + b * imageSizeRGB;
  const T* lap2Image = lap2 + b * imageSizeRGB;
  // The mask is shared (non-batched)
  const T* maskImage = mask;
  T* blendImage = blended + b * imageSizeRGB;

  const F_T F_ONE = static_cast<F_T>(1.0);
  int idx = (y * width + x) * 3;
  F_T m = static_cast<F_T>(maskImage[y * width + x]);
  F_T mm1 = F_ONE - m;
  blendImage[idx + 0] =
      static_cast<T>(m * static_cast<F_T>(lap1Image[idx + 0]) + mm1 * static_cast<F_T>(lap2Image[idx + 0]));
  blendImage[idx + 1] =
      static_cast<T>(m * static_cast<F_T>(lap1Image[idx + 1]) + mm1 * static_cast<F_T>(lap2Image[idx + 1]));
  blendImage[idx + 2] =
      static_cast<T>(m * static_cast<F_T>(lap1Image[idx + 2]) + mm1 * static_cast<F_T>(lap2Image[idx + 2]));
}

// -----------------------------------------------------------------------------
// Batched reconstruction kernel for RGB images.
// Reconstructs the high-resolution image by upsampling the lower-resolution image and
// adding the blended Laplacian details.
// -----------------------------------------------------------------------------
template <typename T, typename F_T>
__global__ void BatchedReconstructKernelRGB(
    const T* __restrict__ lowerRes,
    int lowWidth,
    int lowHeight,
    const T* __restrict__ lap,
    int highWidth,
    int highHeight,
    T* __restrict__ reconstruction,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= highWidth || y >= highHeight)
    return;

  int lowImageSize = lowWidth * lowHeight * 3;
  int highImageSize = highWidth * highHeight * 3;
  const T* lowImage = lowerRes + b * lowImageSize;
  const T* lapImage = lap + b * highImageSize;
  T* reconImage = reconstruction + b * highImageSize;

  const F_T F_ONE = static_cast<F_T>(1.0);
  F_T gx = static_cast<F_T>(x) / 2.0f;
  F_T gy = static_cast<F_T>(y) / 2.0f;
  int gxi = floorf(gx);
  int gyi = floorf(gy);
  F_T dx = gx - static_cast<F_T>(gxi);
  F_T dy = gy - static_cast<F_T>(gyi);
  int gxi1 = min(gxi + 1, lowWidth - 1);
  int gyi1 = min(gyi + 1, lowHeight - 1);

  int idx00 = (gyi * lowWidth + gxi) * 3;
  int idx10 = (gyi * lowWidth + gxi1) * 3;
  int idx01 = (gyi1 * lowWidth + gxi) * 3;
  int idx11 = (gyi1 * lowWidth + gxi1) * 3;

  F_T upR, upG, upB;
  { // Reconstruct R channel.
    F_T val00 = static_cast<F_T>(lowImage[idx00 + 0]);
    F_T val10 = static_cast<F_T>(lowImage[idx10 + 0]);
    F_T val01 = static_cast<F_T>(lowImage[idx01 + 0]);
    F_T val11 = static_cast<F_T>(lowImage[idx11 + 0]);
    F_T val0 = val00 * (F_ONE - dx) + val10 * dx;
    F_T val1 = val01 * (F_ONE - dx) + val11 * dx;
    upR = val0 * (F_ONE - dy) + val1 * dy;
  }
  { // Reconstruct G channel.
    F_T val00 = static_cast<F_T>(lowImage[idx00 + 1]);
    F_T val10 = static_cast<F_T>(lowImage[idx10 + 1]);
    F_T val01 = static_cast<F_T>(lowImage[idx01 + 1]);
    F_T val11 = static_cast<F_T>(lowImage[idx11 + 1]);
    F_T val0 = val00 * (F_ONE - dx) + val10 * dx;
    F_T val1 = val01 * (F_ONE - dx) + val11 * dx;
    upG = val0 * (F_ONE - dy) + val1 * dy;
  }
  { // Reconstruct B channel.
    F_T val00 = static_cast<F_T>(lowImage[idx00 + 2]);
    F_T val10 = static_cast<F_T>(lowImage[idx10 + 2]);
    F_T val01 = static_cast<F_T>(lowImage[idx01 + 2]);
    F_T val11 = static_cast<F_T>(lowImage[idx11 + 2]);
    F_T val0 = val00 * (F_ONE - dx) + val10 * dx;
    F_T val1 = val01 * (F_ONE - dx) + val11 * dx;
    upB = val0 * (F_ONE - dy) + val1 * dy;
  }
  int idxHigh = (y * highWidth + x) * 3;
  reconImage[idxHigh + 0] = static_cast<T>(upR + static_cast<F_T>(lapImage[idxHigh + 0]));
  reconImage[idxHigh + 1] = static_cast<T>(upG + static_cast<F_T>(lapImage[idxHigh + 1]));
  reconImage[idxHigh + 2] = static_cast<T>(upB + static_cast<F_T>(lapImage[idxHigh + 2]));
}

// =============================================================================
// Templated Host Functions: Batched Laplacian Blending
// =============================================================================

template <typename T, typename F_T>
hipError_t cudaBatchedLaplacianBlend(
    const T* h_image1,
    const T* h_image2,
    const T* h_mask,
    T* h_output,
    int imageWidth,
    int imageHeight,
    int numLevels,
    int batchSize,
    hipStream_t stream) {
  // For RGB images (3 channels) and a single-channel mask.
  size_t imageSize = imageWidth * imageHeight * 3 * sizeof(T);
  size_t maskSize = imageWidth * imageHeight * sizeof(T);

  // Allocate device memory for level-0 Gaussian pyramid images.
  std::vector<T*> d_gauss1(numLevels);
  std::vector<T*> d_gauss2(numLevels);
  std::vector<T*> d_maskPyr(numLevels);
  std::vector<T*> d_lap1(numLevels);
  std::vector<T*> d_lap2(numLevels);
  std::vector<T*> d_blend(numLevels);

  std::vector<int> widths(numLevels), heights(numLevels);
  widths[0] = imageWidth;
  heights[0] = imageHeight;
  for (int i = 1; i < numLevels; i++) {
    widths[i] = (widths[i - 1] + 1) / 2;
    heights[i] = (heights[i - 1] + 1) / 2;
  }

  // Allocate level 0 arrays and copy input data from host to device.
  size_t sizeRGB0 = widths[0] * heights[0] * 3 * batchSize * sizeof(T);
  size_t sizeMask0 = widths[0] * heights[0] * sizeof(T);
  hipMalloc((void**)&d_gauss1[0], sizeRGB0);
  hipMalloc((void**)&d_gauss2[0], sizeRGB0);
  hipMalloc((void**)&d_maskPyr[0], sizeMask0);
  hipMemcpyAsync(d_gauss1[0], h_image1, imageSize * batchSize, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_gauss2[0], h_image2, imageSize * batchSize, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_maskPyr[0], h_mask, maskSize, hipMemcpyHostToDevice, stream);

  // Allocate device memory for higher pyramid levels.
  for (int level = 1; level < numLevels; level++) {
    size_t sizeRGB = widths[level] * heights[level] * 3 * batchSize * sizeof(T);
    size_t sizeMask = widths[level] * heights[level] * sizeof(T);
    hipMalloc((void**)&d_gauss1[level], sizeRGB);
    hipMalloc((void**)&d_gauss2[level], sizeRGB);
    hipMalloc((void**)&d_maskPyr[level], sizeMask);
  }

  dim3 block(16, 16, 1);

  // 1. Build Gaussian pyramids for both image sets and the shared mask.
  for (int level = 0; level < numLevels - 1; level++) {
    dim3 gridRGB((widths[level + 1] + block.x - 1) / block.x, (heights[level + 1] + block.y - 1) / block.y, batchSize);
    // Use fused downsampling for the two image sets.
    FusedBatchedDownsampleKernelRGB<T><<<gridRGB, block, 0, stream>>>(
        d_gauss1[level],
        d_gauss2[level],
        widths[level],
        heights[level],
        d_gauss1[level + 1],
        d_gauss2[level + 1],
        widths[level + 1],
        heights[level + 1],
        batchSize);
    // Downsample the shared mask (non-batched).
    {
      dim3 gridMask((widths[level + 1] + block.x - 1) / block.x, (heights[level + 1] + block.y - 1) / block.y, 1);
      BatchedDownsampleKernelMask<T><<<gridMask, block, 0, stream>>>(
          d_maskPyr[level], widths[level], heights[level], d_maskPyr[level + 1], widths[level + 1], heights[level + 1]);
    }
  }

  // 2. Build Laplacian pyramids.
  for (int level = 0; level < numLevels; level++) {
    size_t sizeRGB = widths[level] * heights[level] * 3 * batchSize * sizeof(T);
    hipMalloc((void**)&d_lap1[level], sizeRGB);
    hipMalloc((void**)&d_lap2[level], sizeRGB);
  }
  for (int level = 0; level < numLevels - 1; level++) {
    dim3 grid((widths[level] + block.x - 1) / block.x, (heights[level] + block.y - 1) / block.y, batchSize);
    BatchedComputeLaplacianKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        d_gauss1[level],
        widths[level],
        heights[level],
        d_gauss1[level + 1],
        widths[level + 1],
        heights[level + 1],
        d_lap1[level],
        batchSize);
    BatchedComputeLaplacianKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        d_gauss2[level],
        widths[level],
        heights[level],
        d_gauss2[level + 1],
        widths[level + 1],
        heights[level + 1],
        d_lap2[level],
        batchSize);
  }
  int last = numLevels - 1;
  {
    size_t lastSize = widths[last] * heights[last] * 3 * batchSize * sizeof(T);
    hipMemcpyAsync(d_lap1[last], d_gauss1[last], lastSize, hipMemcpyDeviceToDevice, stream);
    hipMemcpyAsync(d_lap2[last], d_gauss2[last], lastSize, hipMemcpyDeviceToDevice, stream);
  }

  // 3. Blend the Laplacian pyramids using the shared mask.
  for (int level = 0; level < numLevels; level++) {
    size_t sizeRGB = widths[level] * heights[level] * 3 * batchSize * sizeof(T);
    hipMalloc((void**)&d_blend[level], sizeRGB);
    dim3 grid((widths[level] + block.x - 1) / block.x, (heights[level] + block.y - 1) / block.y, batchSize);
    BatchedBlendKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        d_lap1[level], d_lap2[level], d_maskPyr[level], d_blend[level], widths[level], heights[level], batchSize);
  }

  // 4. Reconstruct the final blended image.
  T* d_reconstruct = nullptr;
  hipMalloc((void**)&d_reconstruct, widths[last] * heights[last] * 3 * batchSize * sizeof(T));
  hipMemcpyAsync(
      d_reconstruct,
      d_blend[last],
      widths[last] * heights[last] * 3 * batchSize * sizeof(T),
      hipMemcpyDeviceToDevice,
      stream);
  for (int level = numLevels - 2; level >= 0; level--) {
    T* d_temp = nullptr;
    size_t highSize = widths[level] * heights[level] * 3 * batchSize * sizeof(T);
    hipMalloc((void**)&d_temp, highSize);
    dim3 grid((widths[level] + block.x - 1) / block.x, (heights[level] + block.y - 1) / block.y, batchSize);
    BatchedReconstructKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        d_reconstruct,
        widths[level + 1],
        heights[level + 1],
        d_blend[level],
        widths[level],
        heights[level],
        d_temp,
        batchSize);
    hipFree(d_reconstruct);
    d_reconstruct = d_temp;
  }
  hipMemcpyAsync(h_output, d_reconstruct, imageSize * batchSize, hipMemcpyDeviceToHost, stream);
  hipFree(d_reconstruct);

  // Cleanup allocated device memory.
  for (int level = 0; level < numLevels; level++) {
    hipFree(d_gauss1[level]);
    hipFree(d_gauss2[level]);
    hipFree(d_maskPyr[level]);
    hipFree(d_lap1[level]);
    hipFree(d_lap2[level]);
    hipFree(d_blend[level]);
  }

  return hipGetLastError();
}

template <typename T, typename F_T>
hipError_t cudaBatchedLaplacianBlendWithContext(
    const T* d_image1,
    const T* d_image2,
    const T* d_mask,
    T* d_output,
    CudaBatchLaplacianBlendContext<T>& context,
    hipStream_t stream) {
  size_t imageSize = context.imageWidth * context.imageHeight * 3 * sizeof(T);

  // If context not yet initialized, set up pyramid dimensions and allocate device memory.
  if (!context.initialized) {
    context.widths[0] = context.imageWidth;
    context.heights[0] = context.imageHeight;
    for (int i = 1; i < context.numLevels; i++) {
      context.widths[i] = (context.widths[i - 1] + 1) / 2;
      context.heights[i] = (context.heights[i - 1] + 1) / 2;
      assert(context.widths[i] && context.heights[i]);
    }
    for (int level = 0; level < context.numLevels; level++) {
      size_t sizeRGB = context.widths[level] * context.heights[level] * 3 * context.batchSize * sizeof(T);
      size_t sizeMask = context.widths[level] * context.heights[level] * sizeof(T);
      assert(sizeRGB && sizeMask);
      CUDA_CHECK(hipMalloc((void**)&context.d_lap1[level], sizeRGB));
      context.allocation_size += sizeRGB;
      CUDA_CHECK(hipMalloc((void**)&context.d_lap2[level], sizeRGB));
      context.allocation_size += sizeRGB;
      CUDA_CHECK(hipMalloc((void**)&context.d_blend[level], sizeRGB));
      context.allocation_size += sizeRGB;
      if (level > 0) {
        CUDA_CHECK(hipMalloc((void**)&context.d_maskPyr[level], sizeMask));
        context.allocation_size += sizeMask;
        CUDA_CHECK(hipMalloc((void**)&context.d_gauss1[level], sizeRGB));
        context.allocation_size += sizeRGB;
        CUDA_CHECK(hipMalloc((void**)&context.d_gauss2[level], sizeRGB));
        context.allocation_size += sizeRGB;
      } else {
        context.d_maskPyr[0] = const_cast<T*>(d_mask);
        context.d_gauss1[0] = const_cast<T*>(d_image1);
        context.d_gauss2[0] = const_cast<T*>(d_image2);
      }
    }
  }

  dim3 block(16, 16, 1);

  // 1. Build Gaussian pyramids.
  for (int level = 0; level < context.numLevels - 1; level++) {
    dim3 grid(
        (context.widths[level + 1] + block.x - 1) / block.x,
        (context.heights[level + 1] + block.y - 1) / block.y,
        context.batchSize);
    FusedBatchedDownsampleKernelRGB<T><<<grid, block, 0, stream>>>(
        context.d_gauss1[level],
        context.d_gauss2[level],
        context.widths[level],
        context.heights[level],
        context.d_gauss1[level + 1],
        context.d_gauss2[level + 1],
        context.widths[level + 1],
        context.heights[level + 1],
        context.batchSize);
    CUDA_CHECK(hipGetLastError());
    {
      dim3 gridMask(
          (context.widths[level + 1] + block.x - 1) / block.x, (context.heights[level + 1] + block.y - 1) / block.y, 1);
      BatchedDownsampleKernelMask<T><<<gridMask, block, 0, stream>>>(
          context.d_maskPyr[level],
          context.widths[level],
          context.heights[level],
          context.d_maskPyr[level + 1],
          context.widths[level + 1],
          context.heights[level + 1]);
      CUDA_CHECK(hipGetLastError());
    }
  }

  // 2. Build Laplacian pyramids.
  for (int level = 0; level < context.numLevels - 1; level++) {
    dim3 grid(
        (context.widths[level] + block.x - 1) / block.x,
        (context.heights[level] + block.y - 1) / block.y,
        context.batchSize);
    BatchedComputeLaplacianKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        context.d_gauss1[level],
        context.widths[level],
        context.heights[level],
        context.d_gauss1[level + 1],
        context.widths[level + 1],
        context.heights[level + 1],
        context.d_lap1[level],
        context.batchSize);
    CUDA_CHECK(hipGetLastError());
    BatchedComputeLaplacianKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        context.d_gauss2[level],
        context.widths[level],
        context.heights[level],
        context.d_gauss2[level + 1],
        context.widths[level + 1],
        context.heights[level + 1],
        context.d_lap2[level],
        context.batchSize);
    CUDA_CHECK(hipGetLastError());
  }
  int last = context.numLevels - 1;
  CUDA_CHECK(hipMemcpyAsync(
      context.d_lap1[last],
      context.d_gauss1[last],
      context.widths[last] * context.heights[last] * 3 * sizeof(T) * context.batchSize,
      hipMemcpyDeviceToDevice,
      stream));
  CUDA_CHECK(hipMemcpyAsync(
      context.d_lap2[last],
      context.d_gauss2[last],
      context.widths[last] * context.heights[last] * 3 * sizeof(T) * context.batchSize,
      hipMemcpyDeviceToDevice,
      stream));

  // 3. Blend the Laplacian pyramids.
  for (int level = 0; level < context.numLevels; level++) {
    dim3 grid(
        (context.widths[level] + block.x - 1) / block.x,
        (context.heights[level] + block.y - 1) / block.y,
        context.batchSize);
    BatchedBlendKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        context.d_lap1[level],
        context.d_lap2[level],
        context.d_maskPyr[level],
        context.d_blend[level],
        context.widths[level],
        context.heights[level],
        context.batchSize);
    CUDA_CHECK(hipGetLastError());
  }

  // 4. Reconstruct the final image.
  T* d_reconstruct = nullptr;
  if (!context.initialized) {
    if (context.numLevels > 1) {
      size_t sz = context.widths[last] * context.heights[last] * 3 * sizeof(T) * context.batchSize;
      CUDA_CHECK(hipMalloc((void**)&d_reconstruct, sz));
      context.allocation_size += sz;
      assert(last);
      assert(!context.d_resonstruct[last]);
      context.d_resonstruct[last] = d_reconstruct;
    } else {
      d_reconstruct = d_output;
      assert(last == 0);
      context.d_resonstruct[last] = d_reconstruct;
    }
  } else {
    assert(last >= 0);
    d_reconstruct = context.d_resonstruct[last];
    assert(d_reconstruct);
  }
  CUDA_CHECK(hipMemcpyAsync(
      d_reconstruct,
      context.d_blend[last],
      context.widths[last] * context.heights[last] * 3 * sizeof(T) * context.batchSize,
      hipMemcpyDeviceToDevice,
      stream));
  for (int level = context.numLevels - 2; level >= 0; level--) {
    T* d_temp = nullptr;
    if (!context.initialized) {
      size_t highSize = context.widths[level] * context.heights[level] * 3 * sizeof(T) * context.batchSize;
      if (level > 0) {
        CUDA_CHECK(hipMalloc((void**)&d_temp, highSize));
        context.allocation_size += highSize;
        assert(!context.d_resonstruct[level]);
        context.d_resonstruct[level] = d_temp;
      } else {
        d_temp = d_output;
        assert(highSize == imageSize * context.batchSize);
      }
    } else {
      d_temp = (level > 0) ? context.d_resonstruct[level] : d_output;
    }
    dim3 grid(
        (context.widths[level] + block.x - 1) / block.x,
        (context.heights[level] + block.y - 1) / block.y,
        context.batchSize);
    BatchedReconstructKernelRGB<T, F_T><<<grid, block, 0, stream>>>(
        d_reconstruct,
        context.widths[level + 1],
        context.heights[level + 1],
        context.d_blend[level],
        context.widths[level],
        context.heights[level],
        d_temp,
        context.batchSize);
    CUDA_CHECK(hipGetLastError());
    d_reconstruct = d_temp;
  }
  assert(d_reconstruct == d_output);
  context.initialized = true;

  return hipSuccess;
}

//------------------------------------------------------------------------------
// Explicit template instantiations for supported data types.
//------------------------------------------------------------------------------

#define INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND(T)  \
  template hipError_t cudaBatchedLaplacianBlend<T>( \
      const T* h_image1,                             \
      const T* h_image2,                             \
      const T* h_mask,                               \
      T* h_output,                                   \
      int imageWidth,                                \
      int imageHeight,                               \
      int numLevels,                                 \
      int batchSize,                                 \
      hipStream_t stream);

#define INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND_WITH_CONTEXT(T) \
  template hipError_t cudaBatchedLaplacianBlendWithContext<T>(  \
      const T* d_image1,                                         \
      const T* d_image2,                                         \
      const T* d_mask,                                           \
      T* d_output,                                               \
      CudaBatchLaplacianBlendContext<T>& context,                \
      hipStream_t stream);

INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND(float)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND_WITH_CONTEXT(float)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND(unsigned char)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND_WITH_CONTEXT(unsigned char)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND(__half)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND_WITH_CONTEXT(__half)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND(__hip_bfloat16)
INSTANTIATE_CUDA_BATCHED_LAPLACIAN_BLEND_WITH_CONTEXT(__hip_bfloat16)
