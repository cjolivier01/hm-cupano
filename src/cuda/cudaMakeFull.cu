#include "hip/hip_runtime.h"
#include "cudaMakeFull.h"
#include "cudaTypes.h"

#include "cudaUtils.cuh"

#include <hip/hip_runtime.h>
#include <cassert>

#if (CUDART_VERSION >= 11000)
#include <hip/hip_bf16.h>
#endif
#include <hip/hip_fp16.h>

////////////////////////////////////////////////////////////////////////////////
// Templated Device Kernels
////////////////////////////////////////////////////////////////////////////////

using namespace hm::cupano::cuda;

namespace {

/**
 * @brief Templated batched kernel to copy a region of interest (ROI) from a source image
 *        to a destination canvas while performing a type conversion.
 *
 * For each image in the batch, the kernel copies a rectangular region defined by a source ROI
 * into the destination image. The source pixel values (of type T_in) are converted to type T_out.
 *
 * @tparam T_in  Input pixel type.
 * @tparam T_out Output pixel type.
 * @param src Pointer to the batch of source images in device memory.
 * @param full_src_width Full width of each source image.
 * @param full_src_height Full height of each source image.
 * @param regionWidth Width of the ROI to copy.
 * @param regionHeight Height of the ROI to copy.
 * @param srcROI_x X-coordinate of the top-left corner of the ROI in the source images.
 * @param srcROI_y Y-coordinate of the top-left corner of the ROI in the source images.
 * @param dest Pointer to the batch of destination images in device memory.
 * @param destWidth Width of each destination image.
 * @param destHeight Height of each destination image.
 * @param offsetX X-coordinate in the destination image where the ROI is pasted.
 * @param offsetY Y-coordinate in the destination image where the ROI is pasted.
 * @param batchSize Number of images in the batch.
 */
template <typename T_in, typename T_out>
__global__ void copyRoiKernelBatched(
    const CudaSurface<T_in> src,
    int regionWidth,
    int regionHeight,
    int srcROI_x,
    int srcROI_y,
    CudaSurface<T_out> dest,
    int offsetX,
    int offsetY,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < regionWidth && y < regionHeight) {
    int srcX = srcROI_x + x;
    int srcY = srcROI_y + y;
    if (srcX < src.width && srcY < src.height) {
      int destX = offsetX + x;
      int destY = offsetY + y;
      if (destX < dest.width && destY < dest.height) {
        *surface_ptr(dest, b, destX, destY) = perform_cast<T_out>(*surface_ptr(src, b, srcX, srcY));
      }
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// NEW KERNEL: AlphaConditionalCopyKernel
////////////////////////////////////////////////////////////////////////////////

/**
 * @brief For two surfaces of the same type (which must have an alpha channel),
 *        if image1’s alpha is 0 while image2’s alpha is nonzero, copy image2’s pixel into image1;
 *        and vice versa if image2’s alpha is 0 while image1’s alpha is nonzero.
 *
 * @tparam T The CUDA vector type (e.g. uchar4, float4, half4). It is assumed that T has members .x, .y, .z, and .w.
 * @param image1 The first surface.
 * @param image2 The second surface.
 * @param batchSize Number of images in the batch.
 */
template <typename T>
__global__ void AlphaConditionalCopyKernel(CudaSurface<T> image1, CudaSurface<T> image2, int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= image1.width || y >= image1.height)
    return;

  // Get pointers to the pixel at (x, y) for the current batch element.
  T* pixel1 = surface_ptr(image1, b, x, y);
  T* pixel2 = surface_ptr(image2, b, x, y);

  // If image1's alpha (w) is 0 and image2's is nonzero, copy image2's pixel to image1.
  // Otherwise, if image2's alpha is 0 and image1's is nonzero, copy image1's pixel to image2.
  float alpha1 = static_cast<float>(pixel1->w);
  float alpha2 = static_cast<float>(pixel2->w);
  if (alpha1 == 0.0f && alpha2 != 0.0f) {
    *pixel1 = *pixel2;
  } else if (alpha2 == 0.0f && alpha1 != 0.0f) {
    *pixel2 = *pixel1;
  }
}
} // namespace

/**
 * @brief Launch the AlphaConditionalCopyKernel for surfaces that use a vector type with an alpha channel.
 *
 * @tparam T CUDA vector type (e.g. uchar4, float4, half4).
 * @param image1 Destination/source surface for the first image.
 * @param image2 Destination/source surface for the second image.
 * @param batchSize Number of images in the batch.
 * @param stream CUDA stream to use.
 * @return hipError_t hipGetLastError() after launching the kernel.
 */
template <typename T>
hipError_t AlphaConditionalCopy(CudaSurface<T>& image1, CudaSurface<T>& image2, int batchSize, hipStream_t stream) {
  // Define block and grid dimensions.
  dim3 block(16, 16, 1);
  dim3 grid((image1.width + block.x - 1) / block.x, (image1.height + block.y - 1) / block.y, batchSize);
  // Launch the kernel.
  AlphaConditionalCopyKernel<T><<<grid, block, 0, stream>>>(image1, image2, batchSize);
  return hipGetLastError();
}

#define INSTANTIATE_ALPHA_CONDITIONAL_COPY(T)   \
  template hipError_t AlphaConditionalCopy<T>( \
      CudaSurface<T> & image1, CudaSurface<T> & image2, int batchSize, hipStream_t stream);

INSTANTIATE_ALPHA_CONDITIONAL_COPY(uchar4)
INSTANTIATE_ALPHA_CONDITIONAL_COPY(float4)
INSTANTIATE_ALPHA_CONDITIONAL_COPY(half4)

////////////////////////////////////////////////////////////////////////////////
// Templated Host Functions
////////////////////////////////////////////////////////////////////////////////

/**
 * @brief Creates full canvas images by copying specified source ROIs from a batch of images (and masks)
 *        into preallocated destination canvases.
 *
 * This function fills the destination canvases with default values (for images, 0; for masks, 1)
 * and then copies the ROI from each source image (and source mask) into the corresponding destination canvas.
 *
 * @tparam T Numeric type for images (e.g., float, __half, __hip_bfloat16).
 * @tparam U Numeric type for masks (typically unsigned char).
 * @param d_imgs Pointer to the batch of source images in device memory.
 * @param src_full_width Full width of each source image.
 * @param src_full_height Full height of each source image.
 * @param region_width Width of the ROI to copy from each source image.
 * @param region_height Height of the ROI to copy from each source image.
 * @param d_masks Pointer to the batch of source masks in device memory (or nullptr if not provided).
 * @param mask_width Width of each source mask.
 * @param mask_height Height of each source mask.
 * @param src_roi_x X-coordinate of the top-left corner of the ROI in the source images/masks.
 * @param src_roi_y Y-coordinate of the top-left corner of the ROI in the source images/masks.
 * @param x Reference to destination X-offset for the ROI in the destination canvases (may be adjusted).
 * @param y Reference to destination Y-offset for the ROI in the destination canvases (may be adjusted).
 * @param canvas_w Width of the destination canvases.
 * @param canvas_h Height of the destination canvases.
 * @param batchSize Number of images (and masks) in the batch.
 * @param d_full_imgs Preallocated pointer to the destination canvases for images in device memory.
 * @param d_full_masks Preallocated pointer to the destination canvases for masks in device memory (or nullptr).
 * @param stream CUDA stream to use for kernel launches.
 * @return hipError_t The CUDA error code after kernel launches.
 */
template <typename T_in, typename T_out>
hipError_t simple_make_full_batch(
    const CudaSurface<T_in>& src,
    int region_width,
    int region_height,
    int src_roi_x,
    int src_roi_y,
    int destOffsetX,
    int destOffsetY,
    bool adjust_origin,
    int batchSize,
    CudaSurface<T_out> dest,
    hipStream_t stream) {
  // Ensure the destination offsets are nonnegative.
  assert(destOffsetX >= 0 && destOffsetX >= 0);

  // Define kernel launch parameters.
  dim3 blockDim(16, 16, 1);
  // dim3 gridDimCanvas((dest.width + blockDim.x - 1) / blockDim.x, (dest.height + blockDim.y - 1) / blockDim.y,
  // batchSize);
  dim3 gridDimCanvas(
      (region_width + blockDim.x - 1) / blockDim.x, (region_height + blockDim.y - 1) / blockDim.y, batchSize);

  // -------------------------------------------------------
  // Fill the destination canvases with default values.
  // -------------------------------------------------------
  // For images: fill with 0.
  hipMemsetAsync(dest.d_ptr, 0, total_size(dest, batchSize), stream);

  // -------------------------------------------------------
  // Copy the ROI from each source image/mask into the destination canvases.
  // -------------------------------------------------------
  dim3 gridDimCopy(
      (region_width + blockDim.x - 1) / blockDim.x, (region_height + blockDim.y - 1) / blockDim.y, batchSize);

  // Copy the ROI for the images.
  // Here we use the same type for input and output.
  copyRoiKernelBatched<T_in, T_out><<<gridDimCopy, blockDim, 0, stream>>>(
      src, region_width, region_height, src_roi_x, src_roi_y, dest, destOffsetX, destOffsetY, batchSize);

  return hipGetLastError();
}

/**
 * @brief Interface function for launching the batched ROI copy kernel for images,
 *        with separate input and output types.
 *
 * This function sets up grid and block dimensions and launches the
 * copyRoiKernelBatched kernel.
 *
 * @tparam T_in  Input pixel type.
 * @tparam T_out Output pixel type.
 * @param d_src Pointer to the batch of source images in device memory.
 * @param full_src_width Full width of each source image.
 * @param full_src_height Full height of each source image.
 * @param regionWidth Width of the ROI to copy.
 * @param regionHeight Height of the ROI to copy.
 * @param srcROI_x X-coordinate of the top-left corner of the ROI in the source images.
 * @param srcROI_y Y-coordinate of the top-left corner of the ROI in the source images.
 * @param d_dest Pointer to the batch of destination images in device memory.
 * @param destWidth Width of each destination image.
 * @param destHeight Height of each destination image.
 * @param offsetX X-coordinate in the destination image where the ROI is pasted.
 * @param offsetY Y-coordinate in the destination image where the ROI is pasted.
 * @param batchSize Number of images in the batch.
 * @param stream CUDA stream to use for the kernel launch.
 * @return hipError_t The CUDA error code after kernel launch.
 */
template <typename T_in, typename T_out>
hipError_t copy_roi_batched(
    const CudaSurface<T_in>& src,
    int regionWidth,
    int regionHeight,
    int srcROI_x,
    int srcROI_y,
    CudaSurface<T_out> dest,
    int offsetX,
    int offsetY,
    int batchSize,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((regionWidth + blockDim.x - 1) / blockDim.x, (regionHeight + blockDim.y - 1) / blockDim.y, batchSize);
  copyRoiKernelBatched<T_in, T_out><<<gridDim, blockDim, 0, stream>>>(
      src, regionWidth, regionHeight, srcROI_x, srcROI_y, dest, offsetX, offsetY, batchSize);
  return hipGetLastError();
}

////////////////////////////////////////////////////////////////////////////////
// Explicit Template Instantiations
////////////////////////////////////////////////////////////////////////////////
#define INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(T_in, T_out)     \
  template hipError_t simple_make_full_batch<T_in, T_out>( \
      const CudaSurface<T_in>& src,                         \
      int region_width,                                     \
      int region_height,                                    \
      int src_roi_x,                                        \
      int src_roi_y,                                        \
      int destOffsetX,                                      \
      int destOffsetY,                                      \
      bool adjust_origin,                                   \
      int batchSize,                                        \
      CudaSurface<T_out> dest,                              \
      hipStream_t stream);

#define INSTANTIATE_COPY_ROI_BATCHED(T_in, T_out)     \
  template hipError_t copy_roi_batched<T_in, T_out>( \
      const CudaSurface<T_in>& src,                   \
      int regionWidth,                                \
      int regionHeight,                               \
      int srcROI_x,                                   \
      int srcROI_y,                                   \
      CudaSurface<T_out> dest,                        \
      int offsetX,                                    \
      int offsetY,                                    \
      int batchSize,                                  \
      hipStream_t stream);

// --- Host functions ---
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(uchar3, float3)
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(float3, float3)
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(uchar4, float4)
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(uchar4, float3)
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(uchar4, half3)
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(float4, float4)
// INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(float4, uchar4)
INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(float3, uchar4)
// INSTANTIATE_SIMPLE_MAKE_FULL_BATCH(float4, uchar4)

// Same–type instantiations:
INSTANTIATE_COPY_ROI_BATCHED(half3, uchar3)
INSTANTIATE_COPY_ROI_BATCHED(half3, uchar4)
INSTANTIATE_COPY_ROI_BATCHED(float3, float3)
INSTANTIATE_COPY_ROI_BATCHED(float4, float4)
INSTANTIATE_COPY_ROI_BATCHED(float3, uchar3)
INSTANTIATE_COPY_ROI_BATCHED(float4, uchar4)
INSTANTIATE_COPY_ROI_BATCHED(half4, uchar4)
INSTANTIATE_COPY_ROI_BATCHED(float3, uchar4)
INSTANTIATE_COPY_ROI_BATCHED(uchar3, uchar3)
