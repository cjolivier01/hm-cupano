#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <>
#include "cudaImageAdjust.h"
#include "cudaRemap.h" // Assumed to declare these host functions

namespace {

//------------------------------------------------------------------------------
// Templated Remap Kernel for a Single Image (unchanged)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
__global__ void remapKernel(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX,
    const unsigned short* mapY,
    T_out defR,
    T_out defG,
    T_out defB) {
  // Compute destination pixel coordinates.
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= destW || y >= destH)
    return;

  int destIdx = y * destW + x;

  // Get mapping coordinates and cast them to int.
  int srcX = static_cast<int>(mapX[destIdx]);
  int srcY = static_cast<int>(mapY[destIdx]);

  if (srcX < srcW && srcY < srcH) {
    int srcIdx = (srcY * srcW + srcX) * 3;
    dest[destIdx * 3 + 0] = static_cast<T_out>(src[srcIdx + 0]);
    dest[destIdx * 3 + 1] = static_cast<T_out>(src[srcIdx + 1]);
    dest[destIdx * 3 + 2] = static_cast<T_out>(src[srcIdx + 2]);
  } else {
    dest[destIdx * 3 + 0] = defR;
    dest[destIdx * 3 + 1] = defG;
    dest[destIdx * 3 + 2] = defB;
  }
}

//------------------------------------------------------------------------------
// Templated Batched Remap Kernel for RGB Images (unchanged)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
__global__ void BatchedRemapKernel(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX,
    const unsigned short* mapY,
    T_out defR,
    T_out defG,
    T_out defB,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int srcImageSize = srcW * srcH * 3;
  int destImageSize = destW * destH * 3;
  int mapSize = destW * destH; // mapping arrays match destination size

  const T_in* srcImage = src + b * srcImageSize;
  T_out* destImage = dest + b * destImageSize;
  const unsigned short* mapXImage = mapX + b * mapSize;
  const unsigned short* mapYImage = mapY + b * mapSize;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= destW || y >= destH)
    return;

  int destIdx = y * destW + x;
  int srcX = static_cast<int>(mapXImage[destIdx]);
  int srcY = static_cast<int>(mapYImage[destIdx]);

  if (srcX < srcW && srcY < srcH) {
    int srcIdx = (srcY * srcW + srcX) * 3;
    destImage[destIdx * 3 + 0] = static_cast<T_out>(srcImage[srcIdx + 0]);
    destImage[destIdx * 3 + 1] = static_cast<T_out>(srcImage[srcIdx + 1]);
    destImage[destIdx * 3 + 2] = static_cast<T_out>(srcImage[srcIdx + 2]);
  } else {
    destImage[destIdx * 3 + 0] = defR;
    destImage[destIdx * 3 + 1] = defG;
    destImage[destIdx * 3 + 2] = defB;
  }
}

//------------------------------------------------------------------------------
// Templated Batched Remap Kernel EX (unchanged)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
__global__ void BatchedRemapKernelEx(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX,
    const unsigned short* mapY,
    T_in deflt,
    int batchSize) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int srcImageSize = srcW * srcH;
  int destImageSize = destW * destH;

  const T_in* srcImage = src + b * srcImageSize;
  T_out* destImage = dest + b * destImageSize;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= destW || y >= destH)
    return;

  const int destIdx = y * destW + x;
  const int srcX = static_cast<int>(mapX[destIdx]);
  const int srcY = static_cast<int>(mapY[destIdx]);

  if (srcX < srcW && srcY < srcH) {
    int srcIdx = srcY * srcW + srcX;
    destImage[destIdx] = static_cast<T_out>(srcImage[srcIdx]);
  } else {
    destImage[destIdx] = deflt;
  }
}

//------------------------------------------------------------------------------
// NEW: Templated Batched Remap Kernel for RGB Images with Offset
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
__global__ void BatchedRemapKernelOffset(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX, // mapping arrays of size (remapW x remapH)
    const unsigned short* mapY,
    T_out defR,
    T_out defG,
    T_out defB,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int srcImageSize = srcW * srcH * 3;
  int destImageSize = destW * destH * 3;

  const T_in* srcImage = src + b * srcImageSize;
  T_out* destImage = dest + b * destImageSize;

  // Coordinates within the remap (sub-)region.
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= remapW || y >= remapH)
    return;

  // Compute destination coordinates by adding offset.
  int destX = offsetX + x;
  int destY = offsetY + y;
  if (destX < 0 || destX >= destW || destY < 0 || destY >= destH)
    return;

  int destIdx = destY * destW + destX;
  int mapIdx = y * remapW + x; // mapping arrays are sized remapW x remapH

  int srcX = static_cast<int>(mapX[mapIdx]);
  int srcY = static_cast<int>(mapY[mapIdx]);

  if (srcX < srcW && srcY < srcH) {
    int srcIdx = (srcY * srcW + srcX) * 3;
    destImage[destIdx * 3 + 0] = static_cast<T_out>(srcImage[srcIdx + 0]);
    destImage[destIdx * 3 + 1] = static_cast<T_out>(srcImage[srcIdx + 1]);
    destImage[destIdx * 3 + 2] = static_cast<T_out>(srcImage[srcIdx + 2]);
  } else {
    destImage[destIdx * 3 + 0] = defR;
    destImage[destIdx * 3 + 1] = defG;
    destImage[destIdx * 3 + 2] = defB;
  }
}

//------------------------------------------------------------------------------
// NEW: Templated Batched Remap Kernel EX with Offset (Single-channel)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
__global__ void BatchedRemapKernelExOffset(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX, // mapping arrays of size (remapW x remapH)
    const unsigned short* mapY,
    T_in deflt,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int srcImageSize = srcW * srcH;
  int destImageSize = destW * destH;

  const T_in* srcImage = src + b * srcImageSize;
  T_out* destImage = dest + b * destImageSize;

  // Coordinates within the remap region.
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= remapW || y >= remapH)
    return;

  int destX = offsetX + x;
  int destY = offsetY + y;
  if (destX < 0 || destX >= destW || destY < 0 || destY >= destH)
    return;

  int destIdx = destY * destW + destX;
  int mapIdx = y * remapW + x;

  int srcX = static_cast<int>(mapX[mapIdx]);
  int srcY = static_cast<int>(mapY[mapIdx]);

  if (srcX < srcW && srcY < srcH) {
    int srcIdx = srcY * srcW + srcX;
    destImage[destIdx] = static_cast<T_out>(srcImage[srcIdx]);
  } else {
    destImage[destIdx] = deflt;
  }
}

//------------------------------------------------------------------------------
// NEW: Templated Batched Remap Kernel EX with Offset (Single-channel)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
__global__ void BatchedRemapKernelExOffsetWithDestMap(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX, // mapping arrays of size (remapW x remapH)
    const unsigned short* mapY,
    T_in deflt,
    int this_image_index,
    const unsigned char* dest_image_map,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int srcImageSize = srcW * srcH;
  int destImageSize = destW * destH;

  const T_in* srcImage = src + b * srcImageSize;
  T_out* destImage = dest + b * destImageSize;

  // Coordinates within the remap region.
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= remapW || y >= remapH)
    return;

  int destX = offsetX + x;
  int destY = offsetY + y;
  if (destX < 0 || destX >= destW || destY < 0 || destY >= destH)
    return;

  int destIdx = destY * destW + destX;
  int mapIdx = y * remapW + x;

  int checkIdx = (offsetY + y) * destW + (offsetX + x);

  if (dest_image_map[checkIdx] == this_image_index) {
    int srcX = static_cast<int>(mapX[mapIdx]);
    int srcY = static_cast<int>(mapY[mapIdx]);

    if (srcX < srcW && srcY < srcH) {
      int srcIdx = srcY * srcW + srcX;
      destImage[destIdx] = static_cast<T_out>(srcImage[srcIdx]);
    } else {
      destImage[destIdx] = deflt;
    }
  }
}

template <typename T_in, typename T_out>
__global__ void BatchedRemapKernelExOffsetWithDestMapAdjust(
    const T_in* src,
    int srcW,
    int srcH,
    T_out* dest,
    int destW,
    int destH,
    const unsigned short* mapX, // mapping arrays of size (remapW x remapH)
    const unsigned short* mapY,
    T_in deflt,
    int this_image_index,
    const unsigned char* dest_image_map,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY,
    float3 adjustment) {
  int b = blockIdx.z;
  if (b >= batchSize)
    return;

  int srcImageSize = srcW * srcH;
  int destImageSize = destW * destH;

  const T_in* srcImage = src + b * srcImageSize;
  T_out* destImage = dest + b * destImageSize;

  // Coordinates within the remap region.
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= remapW || y >= remapH)
    return;

  int destX = offsetX + x;
  int destY = offsetY + y;
  if (destX < 0 || destX >= destW || destY < 0 || destY >= destH)
    return;

  int destIdx = destY * destW + destX;
  int mapIdx = y * remapW + x;

  int checkIdx = (offsetY + y) * destW + (offsetX + x);

  if (dest_image_map[checkIdx] == this_image_index) {
    int srcX = static_cast<int>(mapX[mapIdx]);
    int srcY = static_cast<int>(mapY[mapIdx]);
    if (srcX < srcW && srcY < srcH) {
      int srcIdx = srcY * srcW + srcX;
      // Out is more likely to be a float, so adjust after any cast
      destImage[destIdx] = PixelAdjuster<T_out>::adjust(static_cast<T_out>(srcImage[srcIdx]), adjustment);
    } else {
      destImage[destIdx] = deflt;
    }
  }
}

} // anonymous namespace

//------------------------------------------------------------------------------
// Host Function: Remap a Single Image (unchanged)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
hipError_t remap_kernel(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_out defR,
    T_out defG,
    T_out defB,
    hipStream_t stream) {
  dim3 blockDim(16, 16);
  dim3 gridDim((destW + blockDim.x - 1) / blockDim.x, (destH + blockDim.y - 1) / blockDim.y);
  remapKernel<T_in, T_out>
      <<<gridDim, blockDim, 0, stream>>>(d_src, srcW, srcH, d_dest, destW, destH, d_mapX, d_mapY, defR, defG, defB);
  return hipGetLastError();
}

//------------------------------------------------------------------------------
// Host Function: Batched Remap (unchanged)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
hipError_t batched_remap_kernel(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_in defR,
    T_in defG,
    T_in defB,
    int batchSize,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((destW + blockDim.x - 1) / blockDim.x, (destH + blockDim.y - 1) / blockDim.y, batchSize);
  BatchedRemapKernel<T_in, T_out><<<gridDim, blockDim, 0, stream>>>(
      d_src, srcW, srcH, d_dest, destW, destH, d_mapX, d_mapY, defR, defG, defB, batchSize);
  return hipGetLastError();
}

//------------------------------------------------------------------------------
// Host Function: Batched Remap EX (unchanged)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
hipError_t batched_remap_kernel_ex(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_in deflt,
    int batchSize,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((destW + blockDim.x - 1) / blockDim.x, (destH + blockDim.y - 1) / blockDim.y, batchSize);
  BatchedRemapKernelEx<T_in, T_out>
      <<<gridDim, blockDim, 0, stream>>>(d_src, srcW, srcH, d_dest, destW, destH, d_mapX, d_mapY, deflt, batchSize);
  return hipGetLastError();
}

//------------------------------------------------------------------------------
// NEW: Host Function: Batched Remap with Offset (RGB)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
hipError_t batched_remap_kernel_offset(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_in defR,
    T_in defG,
    T_in defB,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((remapW + blockDim.x - 1) / blockDim.x, (remapH + blockDim.y - 1) / blockDim.y, batchSize);
  BatchedRemapKernelOffset<T_in, T_out><<<gridDim, blockDim, 0, stream>>>(
      d_src,
      srcW,
      srcH,
      d_dest,
      destW,
      destH,
      d_mapX,
      d_mapY,
      defR,
      defG,
      defB,
      batchSize,
      remapW,
      remapH,
      offsetX,
      offsetY);
  return hipGetLastError();
}

//------------------------------------------------------------------------------
// NEW: Host Function: Batched Remap EX with Offset (Single-channel)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
hipError_t batched_remap_kernel_ex_offset(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_in deflt,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((remapW + blockDim.x - 1) / blockDim.x, (remapH + blockDim.y - 1) / blockDim.y, batchSize);
  BatchedRemapKernelExOffset<T_in, T_out><<<gridDim, blockDim, 0, stream>>>(
      d_src, srcW, srcH, d_dest, destW, destH, d_mapX, d_mapY, deflt, batchSize, remapW, remapH, offsetX, offsetY);
  return hipGetLastError();
}

//------------------------------------------------------------------------------
// NEW: Host Function: Batched Remap EX with Offset (Single-channel)
//------------------------------------------------------------------------------
template <typename T_in, typename T_out>
hipError_t batched_remap_kernel_ex_offset_with_dest_map(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_in deflt,
    int this_image_index,
    const unsigned char* dest_image_map,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((remapW + blockDim.x - 1) / blockDim.x, (remapH + blockDim.y - 1) / blockDim.y, batchSize);
  BatchedRemapKernelExOffsetWithDestMap<T_in, T_out><<<gridDim, blockDim, 0, stream>>>(
      d_src,
      srcW,
      srcH,
      d_dest,
      destW,
      destH,
      d_mapX,
      d_mapY,
      deflt,
      this_image_index,
      dest_image_map,
      batchSize,
      remapW,
      remapH,
      offsetX,
      offsetY);
  return hipGetLastError();
}

template <typename T_in, typename T_out>
hipError_t batched_remap_kernel_ex_offset_with_dest_map_adjust(
    const T_in* d_src,
    int srcW,
    int srcH,
    T_out* d_dest,
    int destW,
    int destH,
    const unsigned short* d_mapX,
    const unsigned short* d_mapY,
    T_in deflt,
    int this_image_index,
    const unsigned char* dest_image_map,
    int batchSize,
    int remapW,
    int remapH,
    int offsetX,
    int offsetY,
    float3 adjustment,
    hipStream_t stream) {
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((remapW + blockDim.x - 1) / blockDim.x, (remapH + blockDim.y - 1) / blockDim.y, batchSize);
  BatchedRemapKernelExOffsetWithDestMapAdjust<T_in, T_out><<<gridDim, blockDim, 0, stream>>>(
      d_src,
      srcW,
      srcH,
      d_dest,
      destW,
      destH,
      d_mapX,
      d_mapY,
      deflt,
      this_image_index,
      dest_image_map,
      batchSize,
      remapW,
      remapH,
      offsetX,
      offsetY,
      adjustment);
  return hipGetLastError();
}

// Macro for instantiating batched_remap_kernel_offset<T_in, T_out>
#define INSTANTIATE_BATCHED_REMAP_KERNEL_OFFSET(Tin, Tout)     \
  template hipError_t batched_remap_kernel_offset<Tin, Tout>( \
      const Tin* d_src,                                        \
      int srcW,                                                \
      int srcH,                                                \
      Tout* d_dest,                                            \
      int destW,                                               \
      int destH,                                               \
      const unsigned short* d_mapX,                            \
      const unsigned short* d_mapY,                            \
      Tin defR,                                                \
      Tin defG,                                                \
      Tin defB,                                                \
      int batchSize,                                           \
      int remapW,                                              \
      int remapH,                                              \
      int offsetX,                                             \
      int offsetY,                                             \
      hipStream_t stream);

// Macro for instantiating batched_remap_kernel_ex_offset<T_in, T_out>
#define INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET(Tin, Tout)     \
  template hipError_t batched_remap_kernel_ex_offset<Tin, Tout>( \
      const Tin* d_src,                                           \
      int srcW,                                                   \
      int srcH,                                                   \
      Tout* d_dest,                                               \
      int destW,                                                  \
      int destH,                                                  \
      const unsigned short* d_mapX,                               \
      const unsigned short* d_mapY,                               \
      Tin deflt,                                                  \
      int batchSize,                                              \
      int remapW,                                                 \
      int remapH,                                                 \
      int offsetX,                                                \
      int offsetY,                                                \
      hipStream_t stream);

// Macro for instantiating batched_remap_kernel<T_in, T_out>
#define INSTANTIATE_BATCHED_REMAP_KERNEL(Tin, Tout)     \
  template hipError_t batched_remap_kernel<Tin, Tout>( \
      const Tin* d_src,                                 \
      int srcW,                                         \
      int srcH,                                         \
      Tout* d_dest,                                     \
      int destW,                                        \
      int destH,                                        \
      const unsigned short* d_mapX,                     \
      const unsigned short* d_mapY,                     \
      Tin defR,                                         \
      Tin defG,                                         \
      Tin defB,                                         \
      int batchSize,                                    \
      hipStream_t stream);

// Macro for instantiating batched_remap_kernel_ex<T_in, T_out>
#define INSTANTIATE_BATCHED_REMAP_KERNEL_EX(Tin, Tout)     \
  template hipError_t batched_remap_kernel_ex<Tin, Tout>( \
      const Tin* d_src,                                    \
      int srcW,                                            \
      int srcH,                                            \
      Tout* d_dest,                                        \
      int destW,                                           \
      int destH,                                           \
      const unsigned short* d_mapX,                        \
      const unsigned short* d_mapY,                        \
      Tin deflt,                                           \
      int batchSize,                                       \
      hipStream_t stream);

// Macro for instantiating batched_remap_kernel_ex_offset_with_dest_map<T_in, T_out>
#define INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP(Tin, Tout)     \
  template hipError_t batched_remap_kernel_ex_offset_with_dest_map<Tin, Tout>( \
      const Tin* d_src,                                                         \
      int srcW,                                                                 \
      int srcH,                                                                 \
      Tout* d_dest,                                                             \
      int destW,                                                                \
      int destH,                                                                \
      const unsigned short* d_mapX,                                             \
      const unsigned short* d_mapY,                                             \
      Tin deflt,                                                                \
      int this_image_index,                                                     \
      const unsigned char* dest_image_map,                                      \
      int batchSize,                                                            \
      int remapW,                                                               \
      int remapH,                                                               \
      int offsetX,                                                              \
      int offsetY,                                                              \
      hipStream_t stream);

// Macro for instantiating batched_remap_kernel_ex_offset_with_dest_map_adjust<T_in, T_out>
#define INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP_ADJUST(Tin, Tout)     \
  template hipError_t batched_remap_kernel_ex_offset_with_dest_map_adjust<Tin, Tout>( \
      const Tin* d_src,                                                                \
      int srcW,                                                                        \
      int srcH,                                                                        \
      Tout* d_dest,                                                                    \
      int destW,                                                                       \
      int destH,                                                                       \
      const unsigned short* d_mapX,                                                    \
      const unsigned short* d_mapY,                                                    \
      Tin deflt,                                                                       \
      int this_image_index,                                                            \
      const unsigned char* dest_image_map,                                             \
      int batchSize,                                                                   \
      int remapW,                                                                      \
      int remapH,                                                                      \
      int offsetX,                                                                     \
      int offsetY,                                                                     \
      float3 adjustment,                                                               \
      hipStream_t stream);

// For batched_remap_kernel_offset
INSTANTIATE_BATCHED_REMAP_KERNEL_OFFSET(float, float)
INSTANTIATE_BATCHED_REMAP_KERNEL_OFFSET(uchar1, uchar1)
INSTANTIATE_BATCHED_REMAP_KERNEL_OFFSET(float, __half)
INSTANTIATE_BATCHED_REMAP_KERNEL_OFFSET(__half, float)
INSTANTIATE_BATCHED_REMAP_KERNEL_OFFSET(__half, __half)

// For batched_remap_kernel_ex_offset
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET(float3, float3)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET(uchar3, uchar3)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET(float, float)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET(__half, __half)

// For batched_remap_kernel
INSTANTIATE_BATCHED_REMAP_KERNEL(float, float)
INSTANTIATE_BATCHED_REMAP_KERNEL(float, __half)
INSTANTIATE_BATCHED_REMAP_KERNEL(__half, float)
INSTANTIATE_BATCHED_REMAP_KERNEL(__half, __half)
INSTANTIATE_BATCHED_REMAP_KERNEL(uchar1, uchar1)

// For batched_remap_kernel_ex
INSTANTIATE_BATCHED_REMAP_KERNEL_EX(float3, float3)

// Instantiate for float input and float output:
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP(float, float)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP(float3, float3)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP(uchar1, uchar1)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP(uchar3, uchar3)

INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP_ADJUST(float3, float3)
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP_ADJUST(uchar3, uchar3)

// Instantiate for __half input and __half output:
INSTANTIATE_BATCHED_REMAP_KERNEL_EX_OFFSET_WITH_DEST_MAP(__half, __half)
